#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <windows.h>

// OpenGL Graphics includes
#include <GL/glew.h>

//#include <GL/freeglut.h>
#include <GL/glut.h>

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

//threadIdx vs. için
#include ""

#include "arac.h"
#include "main.h"
#include "Harita.h"



//render fonksiyonlarý

void RenderScene();
void waitAndRedraw(int deger);

//CUDA fonksiyonlarý
void launch_kernel_Update(); //kernel çaðýran fonksiyon


////////////////////////////////////////////////////////////


ARAC *araclar[arac_sayisi];



float RBP=0.03;
float toplam=0.f;







////////////////////////////////////////////////////////////
float SOLX[arac_sayisi];
float SOLY[arac_sayisi];
int TIP[arac_sayisi];
int HIZ[arac_sayisi];
int YON[arac_sayisi];
int YOLID[arac_sayisi];
int SERITID[arac_sayisi];
int HCID[arac_sayisi];

////////////////////////////////////////////////////////////


////////////////////////////////////////////////////
////////////////////////////////////////////////////
void HucreDoldur(int y_id,int s_id,int h_id,int tip)//en baþta dolu olacak hücreler belirleniyor
{
	int kap_hc_say;
	if(tip==3 || tip==1)
	{
		kap_hc_say=2;
	}
	else
		kap_hc_say=1;
	int count=h_id;
	if(y_id==1)
	{
		for(int j=0;j<kap_hc_say;j++)
		{
			yol1[count][s_id]=1;
			count++;
		}
	}
	if(y_id==2)
	{
		for(int j=0;j<kap_hc_say;j++)
		{
			yol2[count][s_id]=1;
			count++;
		}
	}
		if(y_id==3)
	{
		for(int j=0;j<kap_hc_say;j++)
		{
			yol3[count][s_id]=1;
			count++;
		}
	}
	if(y_id==5)
	{
		for(int j=0;j<kap_hc_say;j++)
		{
			yol5[count][s_id]=1;
			count++;
		}
	}

	if(y_id==7)
	{
		for(int j=0;j<kap_hc_say;j++)
		{
			yol7[count][s_id]=1;
			count++;
		}
		
	}
	if(y_id==8)
	{
		for(int j=0;j<kap_hc_say;j++)
		{
			yol8[count][s_id]=1;
			count++;
		}
	}
	if(y_id==9)
	{
		for(int j=0;j<kap_hc_say;j++)
		{
			yol9[count][s_id]=1;
			count++;
		}
	}
	if(y_id==10)
	{
		for(int j=0;j<kap_hc_say;j++)
		{
			yol10[count][s_id]=1;
			count++;
		}
	}
	
	if(y_id==52)
	{
		for(int j=0;j<kap_hc_say;j++)
		{
			yol52[count][s_id]=1;
			count++;
		}
	}
	
}
///////////////////////////

//////////////////////////////////
int getForwardGap(int y_id,int s_id,int h_id, int tip)
{
	int gap=0;
	int count;
	if(tip==3 || tip==1)
		count=2;
	else
		count=1;
	if(y_id==8)
	{
		
		while(  yol8[h_id+count][s_id]==0 && h_id+count<90)
		{
			
			count++;
			gap++;
			
			
		}
		
		
		return gap;
	}

	
}
///////////////////////////////////////////////////
__global__ void kernel(float *sx,float *sy,int *tip, int *hiz,int *yon,int *yolID,int *srtID,int *hcID,int *yol1,int *yol2,int *yol3,int *yol5, int *yol52, int *yol7 ,int *yol8,int *yol9,int* yol10, int *TrafficLight,int *kol1,int *kol2, int *kol3, int* kol4,int *kol2_1,int *kol2_2,int *kol2_3, int *kol2_4)
{
	int id=threadIdx.x + blockIdx.x * blockDim.x;
	
	/////////////////////////////////////////
	////YOL1
		if(yolID[id]==1 )//yol1 mi?
		{
			int hdx=hcID[id];
			int sdx=srtID[id];
			int serit_sayi=2;
			int idx=hdx*serit_sayi+sdx;

			//kavþak noktasýna kadar ilerle
			if(sdx==0 && yon[id]==1)
			{
				
				if(yol1[idx+serit_sayi]==0)
				{   
					sx[id]+=7.5;
					yol1[idx+serit_sayi]=1;
					yol1[idx]=0;
					hcID[id]++;
				}
			}
			////saða döndükten sonra aþaðý yönde ilerle
			if(sdx==0 && yon[id]==4)
			{
				if(yol1[idx+1*serit_sayi]==0)
				{   
					sy[id]-=7.5;
					yol1[idx+1*serit_sayi]=1;
					yol1[idx]=0;
					hcID[id]++;
				}
			}
			//yol1-þerit1-yön 3(yukarý gidiþ)
			if(sdx==1 && yon[id]==3)
			{
				if(yol1[idx-1*serit_sayi]==0)
				{   
					sy[id]+=7.5;
					yol1[idx-1*serit_sayi]=1;
					yol1[idx]=0;
					hcID[id]--;
				}
			}
			////yol1-þerit1-sola dönüþ
			if(hdx==19 && sdx==1 ){
					sx[id]=36*3.75f;
					sy[id]=85*3.75;
					yon[id]=2;
					yolID[id]=1;
					srtID[id]=1;
					hcID[id]=18;
					yol1[19*serit_sayi+1]=0;
					
					
			}
			////þerit 1--->çýkýþ noktasýna kadar ilerle
			if(sdx==1 && yon[id]==2)
			{
				
				if(yol1[idx-1*serit_sayi]==0)
				{   
					sx[id]-=7.5;
					yol1[idx-1*serit_sayi]=1;
					yol1[idx]=0;
					hcID[id]--;
				}
			}
			////yol1-þerit1-haritadan çýkýþ--->yol5 giriþ
			if(hdx==0 && sdx==1 )// hedef hc boþ mu?
			{
					sx[id]=95*3.75f;
					sy[id]=91*3.75;
					yon[id]=4;
					yolID[id]=5;
					srtID[id]=0;
					yol1[0*serit_sayi+1]=0;
					hcID[id]=1;
					
					
			}
			////yol1&serit0(saða dönüþ)
			if(hdx==19 && sdx==0 )// gideceði hc. boþ mu?
			{
					sx[id]=36*3.75f;
					sy[id]=83*3.75;
					yon[id]=4;
					yolID[id]=1;
					srtID[id]=0;
					hcID[id]=20;
					yol1[19*serit_sayi+0]=0;
			}
			////yol1&serit0(saða dönüþ-otoyola çýkýþ)
			if(hdx==34 && sdx==0 && yol7[18*serit_sayi+0]==0) //hedef hc boþ mu?
			{
					sx[id]=38*3.75f;
					sy[id]=53*3.75;
					yon[id]=2;
					yolID[id]=7;
					srtID[id]=0;
					yol1[34*serit_sayi+0]=0;
					hcID[id]=18;
			}
		
		}//end of YOL1
	////////////////////////////////////////////////////////////////////////////////////////////
		//YOL2
	if(yolID[id]==2 )
		{
			int hdx=hcID[id];
			int sdx=srtID[id];
			int serit_sayi=2;
			int idx=hdx*serit_sayi+sdx;

			//kavþak noktasýna kadar ilerle
			if(sdx==0 && yon[id]==1)
			{
				
				if(yol2[idx+1*serit_sayi]==0)
				{   
					sx[id]+=7.5;
					yol2[idx+1*serit_sayi]=1;
					yol2[idx]=0;
					hcID[id]++;
				}
			}
			//þerit 1 yön 2
			if(sdx==1 && yon[id]==2)
			{
				
				if(yol2[idx-1*serit_sayi]==0)
				{   
					sx[id]-=7.5;
					yol2[idx-1*serit_sayi]=1;
					yol2[idx]=0;
					hcID[id]--;
				}
			}
			//þerit 2 yol sonu(baþladýðý konuma geri dönecek)
			if(sdx==1 && hcID[id]==0)
			{
					sx[id]=138*3.75;
					sy[id]=90*3.75;
					yon[id]=4;
					yolID[id]=10;
					srtID[id]=1;
					yol2[0*serit_sayi+1]=0;
					hcID[id]=42;
			}

			//yol2 sonuna gelindi, yukarý dönecek
			if(sdx==0 && hcID[id]==72 )
			{
				
						if(yol10[1*serit_sayi+0]==0)
						{
							sx[id]=141*3.75;
							sy[id]=6*3.75;
							yon[id]=3;
							srtID[id]=0;
							yolID[id]=10;
							hcID[id]=1;
							yol2[72*serit_sayi+0]=0;
					    }
			}	
			
			
			
		}//end of YOL2
	
		////////////////////////////////////////////////////////////////////////////////////////////
		//YOL3
		if(yolID[id]==3 )
		{
			int hdx=hcID[id];
			int sdx=srtID[id];
			int serit_sayi=2,
			int idx=hdx*serit_sayi+sdx;

			//kavþak noktasýna kadar ilerle
			if(sdx==0 && yon[id]==3)
			{
				
				if(yol3[idx+1*serit_sayi]==0)
				{   
					sy[id]+=7.5;
					yol3[idx+1*serit_sayi]=1;
					yol3[idx]=0;
					hcID[id]++;
				}
			}

			
			//yol3 sonuna gelindi, saða dönecek--->YOL8
			if(sdx==0 && hdx==20)
			{
				sx[id]=38*3.75;
				sy[id]=46*3.75;
				yon[id]=1;
				srtID[id]=0;
				yolID[id]=8;
				yol3[20*serit_sayi+0]=0;
				hcID[id]=20;//yol8'un 19.hücresi
				yol8[20*serit_sayi+0]=true;
				
			}
			//yol3 þerit1 aþaðý yönde ilerle
			if(sdx==1 && yon[id]==4)
			{
				
				if(yol3[idx-1*serit_sayi]==0)
				{   
					sy[id]-=7.5;
					yol3[idx-1*serit_sayi]=1;
					yol3[idx]=0;
					hcID[id]--;
				}
			}
			//yol3---->yol2 geçiþ saða dönüþ
			if(sdx==1 && hcID[id]==0)
			{
				sx[id]=36*3.75;
				sy[id]=6*3.75;
				yon[id]=2;
				srtID[id]=1;
				yolID[id]=2;
				yol3[0*serit_sayi+1]=0;
				hcID[id]=18;
			}
		}//end of YOL3

		////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////
		//YOL5
		if(yolID[id]==5 )
		{
			int hdx=hcID[id];
			int sdx=srtID[id];
			int serit_sayi=2;
			int idx=hdx*serit_sayi+sdx;

			//kavþak noktasýna kadar ilerle
			if(sdx==0 && yon[id]==4)
			{
				
				if(yol5[idx+1*serit_sayi]==0)
				{   
					sy[id]-=7.5;
					yol5[idx+1*serit_sayi]=1;
					yol5[idx]=0;
					hcID[id]++;
				}

				if(hdx>=19 && hdx<25)
				{
					sx[id]=1500.f;
					yol5[20*serit_sayi+0]=0;
					hcID[id]++;
				}
				if(hdx==25)
				{
					sx[id]=95*3.75;
					sy[id]=46*3.75;
					
				}
				//yol2'ye dönüþ
				
				if(hdx==42   )
				{
					
						sx[id]=94*3.75;
						sy[id]=6*3.75;
						yolID[id]=2;
						srtID[id]=1,
						hcID[id]=48;
						yol2[48*serit_sayi+1]=1;
						yol5[43*serit_sayi+0]=0;
						yon[id]=2;
					
				}
				
			}
			if(sdx==1 && yon[id]==3)
			{
				
				if(yol5[idx-1*serit_sayi]==0)
				{   
					sy[id]+=7.5;
					yol5[idx-1*serit_sayi]=1;
					yol5[idx]=0;
					hcID[id]--;
				}

				if(hdx<=23 && hdx>19)
				{
					sx[id]=1500.f;
					yol5[22*serit_sayi+1]=0;
					hcID[id]--;
				}
				if(hdx==19)
				{
					sx[id]=97*3.75;
					sy[id]=53*3.75;
					
					
				}
				//yol5------------>çýkýþ
				if(sdx==1 && hdx==0)
				{
					//yol1 serit0'a gönder
					sx[id]=0*3.75;
					sy[id]=83*3.75;
					yolID[id]=1;
					srtID[id]=0;
					hcID[id]=1;
					yon[id]=1;
					yol1[1*serit_sayi+0]=1;
					yol5[0*serit_sayi+1]=0;
				}
			}
			
		
			

			
			
		}//end of YOL5
		////////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////////////////////////
		//YOL52
		if(yolID[id]==52 )
		{
			int hdx=hcID[id];
			int sdx=srtID[id];
			int serit_sayi=2;
			int idx=hdx*serit_sayi+sdx;
			//þerit 1- þerit sonuna kadar ilerle
			if(sdx==1 && yon[id]==4)
			{
				
				if(yol52[idx-1*serit_sayi]==0)
				{   
					sy[id]-=7.5;
					yol52[idx-1*serit_sayi]=1;
					yol52[idx]=0;
					hcID[id]--;
				}
			}
			//yol52 yukarý yönde ilerle
			if(sdx==0 && yon[id]==3)
			{
				
				if(yol52[idx+1*serit_sayi]==0)
				{   
					sy[id]+=7.5;
					yol52[idx+1*serit_sayi]=1;
					yol52[idx]=0;
					hcID[id]++;
				}
			}
			
			if(hdx==0 && yol2[47*serit_sayi+1]==0)//gideceði hücre boþ mu?
					{
						//saða dön---->yol2
						sx[id]=94*3.75f;
						sy[id]=6*3.75;
						yon[id]=2;
						yolID[id]=2;
						srtID[id]=1;
						yol52[0*serit_sayi+1]=0;
						hcID[id]=47;

					}
			//çapraz yol(5 yönünde)
			if(hdx==15 && sdx==0)
					{
						
						sx[id]=98*3.75f;
						sy[id]=37*3.75;
						yon[id]=5;
						yolID[id]=104;
						srtID[id]=1;
						hcID[id]=0;
						yol52[idx]=0;
					}
					
		}//end of YOL52
		///////////////////////////////////////////////////////////////
		//YOL7
	if(yolID[id]==7)
	{
		int hdx=hcID[id];
		int sdx=srtID[id];
		int serit_sayi=3;
		if(tip[id]==0 || tip[id]==2 || tip[id]==4)
				{
					//ýþýklara kadar devam et
					if(hdx>72 && yol7[hdx*serit_sayi+sdx-serit_sayi]==0 )
					 {
							sx[id]-=7.5;
							yol7[hdx*serit_sayi+sdx-serit_sayi*hiz[id]]=1;
							yol7[hdx*serit_sayi+sdx]=0;
							hcID[id]--;
					 }
				}
			//	//kýrmýzý yandý,dur
				if(TrafficLight[0]==0 && hdx==72)
				 {
					 sx[id]-=(hiz[id]*0.f);
				 }
			//	 //yeþil yandý, geç
				if(TrafficLight[0]==2 && hdx==72 && yol7[hdx*serit_sayi+sdx-serit_sayi]==0)
				 {
							sx[id]-=7.5;
							yol7[hdx*serit_sayi+sdx-serit_sayi*hiz[id]]=1;
							yol7[hdx*serit_sayi+sdx]=0;
							hcID[id]--;
				 }
			//	 //ýþýklarý geçtikten sonra durma ,devam et
				if(hdx<=71 && yol7[hdx*serit_sayi+sdx-serit_sayi]==0)
				 {					
							
							sx[id]-=7.5;
							yol7[hdx*serit_sayi+sdx-serit_sayi*hiz[id]]=1;
							yol7[hdx*serit_sayi+sdx]=0;
							hcID[id]--;
				 }

				//çapraz yol
				 if(sdx==0 && hdx==73)
				 {
						sx[id]=146.5*3.75f;
						sy[id]=52*3.75;
						yon[id]=8;
						yolID[id]=202;
						srtID[id]=1;
						hcID[id]=0;
						yol7[73*3+0]=0;

				 }
				//haritadan çýkýþlar
			if(hdx==1 && sdx==0)
			{
				sx[id]=180*3.75;
				sy[id]=53*3.75;
				yon[id]=2;
				yolID[id]=7;
				srtID[id]=0;
				yol7[1*serit_sayi+0]=0;
				hcID[id]=90;
				
			}
			if(hdx==1 && sdx==1)
			{
				sx[id]=180*3.75;
				sy[id]=52*3.75;
				yon[id]=2;
				yolID[id]=7;
				srtID[id]=1;
				yol7[1*serit_sayi+1]=0;
				hcID[id]=90;
				
			}
			if(hdx==1 && sdx==2)//yol7&serit2(çýkýþ)
			{
				sx[id]=180*3.75;
				sy[id]=51*3.75;
				yon[id]=2;
				yolID[id]=7;
				srtID[id]=2;
				yol7[1*serit_sayi+2]=0;
				hcID[id]=90;
			}


	}
	/////////////////////////////////////////
	if(yolID[id]== 8 )
		{		
			int hdx=hcID[id];
			int sdx=srtID[id];
			int serit_sayi=3;
			int idx=hdx*serit_sayi+sdx;
			//int vmax;
			//	//yol8(Bölünmüþ ana yol) için max. hýzlar
			//	if(tip[id]==0)
			//		vmax=4;//otomobil vmax=108km/sa
			//	else if(tip[id]==1)
			//		vmax=2;//týr vmax=54km/sa
			//	else if (srtID[id]==0)
			//		vmax=1;
			//		
			//	else 
			//		vmax=3;//kamyon,kamyonet,otobüs vmax=81km/sa
			//	//Nagel-Sch. Modeli

			//	//Adým 1: Hýzlanma(Acceleration)
			//	if(hiz[id] < vmax)
			//		hiz[id]++;
			//	

			//	//Adým 2: Yavaþlama (Braking)
			//	//hiz[id]=minimum(hiz[id],getForwardGap(yolID[id],srtID[id],hcID[id],tip[id]));
			//	
			//	////Adým 3: Random Braking
			//	//float x=float(rand())/RAND_MAX;
			//
			//	//if(x<=RBP)
			//	//	{
			//	//		araclar[id]->hiz=maximum(0,araclar[id]->hiz-1);
			//	//		
			//	//	}
			//	////Adým 4:Move(aþaðýda kodlarý mevcut)

			//	////ýþýklara yaklaþýnca yavaþla
			//	
			//	if(hcID[id]>=60 )
			//		hiz[id]=1;

			////araç tipi týr ya da otobüs ise(yani 2 hc. kaplýyorsa)
				if(tip[id]==3 || tip[id]==1)
			    {
				//ýþýklara kadar devam et
				if(hdx+serit_sayi<=67 && yol8[idx+serit_sayi*2]==0 )
				 {
							sx[id]+=7.5;
							yol8[idx+serit_sayi]=1;
							yol8[idx+serit_sayi*2]=1;

							yol8[idx]=0;
							yol8[idx-1*serit_sayi]=0;
							hcID[id]++;
				 }
				}
			
				//kýrmýzý yandý,dur
				if(TrafficLight[2]==0 && (hdx+1)==68)
				 {
					 sx[id]+=0.f;
				 }
				 //yeþil yandý, geç
				if(TrafficLight[2]==2 && (hdx+1)==68 && yol8[hdx*serit_sayi+sdx+serit_sayi*2]==0)
				 {
							sx[id]+=7.5;
							yol8[idx+serit_sayi]=1;
							yol8[idx]=0;
							hcID[id]++;
				 }

				 //ýþýklarý geçtikten sonra durma ,devam et
				if(hdx>=69 && yol8[hdx*serit_sayi+sdx+serit_sayi*2]==0)
				 {					
							sx[id]+=7.5;
							yol8[idx+serit_sayi]=1;
							yol8[idx]=0;
							hcID[id]++;
				 }

			/////////////////////////////////////////////////
			//tek hücreliler :)

				if(tip[id]==0 || tip[id]==2 || tip[id]==4)
				{
					//ýþýklara kadar devam et
					if(hdx<=67 && yol8[hdx*serit_sayi+sdx+serit_sayi]==0 )
					 {
							sx[id]+=7.5;
							yol8[hdx*serit_sayi+sdx+serit_sayi*hiz[id]]=1;
							yol8[hdx*serit_sayi+sdx]=0;
							hcID[id]++;
					 }
				}
			//	//kýrmýzý yandý,dur
				if(TrafficLight[2]==0 && hdx==68)
				 {
					 sx[id]+=(hiz[id]*0.f);
				 }
			//	 //yeþil yandý, geç
				if(TrafficLight[2]==2 && hdx==68 && yol8[hdx*serit_sayi+sdx+serit_sayi]==0)
				 {
							sx[id]+=7.5;
							yol8[hdx*serit_sayi+sdx+serit_sayi*hiz[id]]=1;
							yol8[hdx*serit_sayi+sdx]=0;
							hcID[id]++;
				 }
			//	 //ýþýklarý geçtikten sonra durma ,devam et
				if(hdx>=69 && yol8[hdx*serit_sayi+sdx+serit_sayi]==0)
				 {					
							
							sx[id]+=7.5;
							yol8[hdx*serit_sayi+sdx+serit_sayi*hiz[id]]=1;
							yol8[hdx*serit_sayi+sdx]=0;
							hcID[id]++;
				 }
			//	 //////////////////////////////////////////////////////////////////////////////////////////////

			//	 if(araclar[id]->serit_id==0 && araclar[id]->hc_id==18)
			//	 {
			//		int tempYon=rand()%2+1;

			//		if (tempYon==1)
			//		{
			//			yon=1;
			//		}
			//		else
			//		{
			//			if(yol3[20][1]==false )
			//			{
			//				araclar[id]->solX=36*3.75;
			//				araclar[id]->solY=46*3.75;
			//				araclar[id]->yon=4;
			//				araclar[id]->serit_id=1;
			//				araclar[id]->yol_id=3;
			//				araclar[id]->hc_id=20;
			//				yol8[18][0]=false;
			//			}
			//		} 
			//	 }
			//	  if(araclar[id]->serit_id==0 && araclar[id]->hc_id==31)
			//	 {
			//		int tempYon=rand()%2+1;

			//		if (tempYon==1)
			//		{
			//			yon=1;
			//		}
			//		else
			//		{
			//			if(yol16[8][1]==false )
			//			{
			//				araclar[id]->solX=61*3.75;
			//				araclar[id]->solY=46*3.75;
			//				araclar[id]->yon=4;
			//				araclar[id]->serit_id=1;
			//				araclar[id]->yol_id=16;
			//				araclar[id]->hc_id=8;
			//				yol8[31][0]=false;
			//			}
			//		} 
			//	 }
			//	   if(araclar[id]->serit_id==0 && araclar[id]->hc_id==43)
			//	 {
			//		int tempYon=rand()%2+1;

			//		if (tempYon==1)
			//		{
			//			yon=1;
			//		}
			//		else
			//		{
			//			if(kol3[0]==false )
			//			{
			//				araclar[id]->solX=86*3.75;
			//				araclar[id]->solY=45*3.75;
			//				araclar[id]->yon=6;
			//				
			//				araclar[id]->yol_id=103;
			//				araclar[id]->hc_id=0;
			//				yol8[43][0]=false;
			//			}
			//		} 
			//	 }
			//	  if(araclar[id]->serit_id==0 && araclar[id]->hc_id==67)
			//	 {
			//		
			//			if(kol2_3[0]==false )
			//			{
			//				araclar[id]->solX=134*3.75;
			//				araclar[id]->solY=45*3.75;
			//				araclar[id]->yon=6;
			//				
			//				araclar[id]->yol_id=203;
			//				araclar[id]->hc_id=0;
			//				yol8[67][0]=false;
			//			}
			//	 } 
			//	 
			////haritadan çýkýþlar
			if(hdx==90 && sdx==0)//yol8&serit0(çýkýþ)
			{
				sx[id]=0*3.75;
				sy[id]=46*3.75;
				yon[id]=1;
				yolID[id]=8;
				srtID[id]=0;
				yol8[90*serit_sayi+0]=0;
				hcID[id]=1;
			}

			if(hdx==90 && sdx==1)
			{
				sx[id]=0*3.75;
				sy[id]=47*3.75;
				yon[id]=1;
				yolID[id]=8;
				srtID[id]=1;
				yol8[90*serit_sayi+1]=0;
				hcID[id]=1;
				
			}
			if(hdx==90 && sdx==2)//yol8&serit2(çýkýþ)
			{
				sx[id]=0*3.75;
				sy[id]=48*3.75;
				yon[id]=1;
				yolID[id]=8;
				srtID[id]=2;
				yol8[90*serit_sayi+2]=0;
				hcID[id]=1;
			}
		
			//
			//
		}//end  of YOL8
		///////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////
		//YOL9
		if(yolID[id]==9 )
		{
			int hdx=hcID[id];
			int sdx=srtID[id];
			int serit_sayi=2;
			int idx=hdx*serit_sayi+sdx;
			//þerit 1- þerit sonuna kadar ilerle
			if(sdx==1 )
			{
				
				if(yol9[idx-1*serit_sayi]==0)
				{   
					sy[id]-=7.5;
					yol9[idx-1*serit_sayi]=1;
					yol9[idx]=0;
					hcID[id]--;
				}
				
				if(hdx==0 && yol2[69*serit_sayi+1]==0)//gideceði hücre boþ mu?
					{
						//saða dön---->yol2
						
						sx[id]=136*3.75f;
						sy[id]=6*3.75;
						yon[id]=2;
						yolID[id]=2;
						srtID[id]=1;
						yol9[0*2+1]=0;
						hcID[id]=69;

					}
			} //serit1 sonu

			
			//yol9 yukarý yönde ilerle
			if(sdx==0 && yon[id]==3)
			{
				
				if(yol9[idx+1*serit_sayi]==0)
				{   
					sy[id]+=7.5;
					yol9[idx+1*serit_sayi]=1;
					yol9[idx]=0;
					hcID[id]++;
				}
				if(hdx==19 && sdx==0)
				{
						sx[id]=141*3.75f;
						sy[id]=41*3.75;
						yon[id]=5;
						yolID[id]=204;
						srtID[id]=1;
						hcID[id]=0;
						yol9[19*serit_sayi+0]=0;
				}
			}
			
			
			
					
		}//end of YOL9
		///////////////////////////////////////////////////////////////////////////////
		//YOL10
		if(yolID[id]==10 )
		{
			int hdx=hcID[id];
			int sdx=srtID[id];
			int serit_sayi=2;
			int idx=hdx*serit_sayi+sdx;
			//serit0 ----->yon 3(yani yukarý)
			if(sdx==0)
			{
				//ýþýklara kadar devam et
				if(hdx<=19 && yol10[idx+1*serit_sayi]==0 )
				 {
							sy[id]+=(hiz[id]*7.5);
							yol10[idx+hiz[id]*serit_sayi]=1;
							yol10[idx]=0;
							hcID[id]+=(hiz[id]);
				 }
		
				//kýrmýzý yandý,dur
				if(TrafficLight[1]==0 && hdx==20)
				 {
					 sy[id]+=(hiz[id]*0.f);
				 }
				 //yeþil yandý, geç
				if(TrafficLight[1]==2 && hdx==20 && yol10[idx+1*serit_sayi]==0)
				 {
							sy[id]+=(hiz[id]*7.5);
							yol10[idx+hiz[id]*serit_sayi]=1;
							yol10[idx]=0;
							hcID[id]+=(hiz[id]);
				 }
				 //ýþýklarý geçtikten sonra durma ,devam et
				if(hdx>=21 && yol10[idx+1*serit_sayi]==0)
				 {					
							sy[id]+=(hiz[id]*7.5);
							yol10[idx+hiz[id]*serit_sayi]=1;
							yol10[idx]=0;
							hcID[id]+=(hiz[id]);
				 }

				//yol10-serit0 sonu----->yol2'ye dön
				if(hdx== 42)
				{
					
					sx[id]=0.f;
					sy[id]=4*3.75;
					yolID[id]=2,
					srtID[id]=0;
					hcID[id]=1;
					yol10[42*serit_sayi+0]=0;
					yol2[1*serit_sayi+0]=1;
					yon[id]=1;
				}
			
			}
			//serit1----->yon4 (yani aþaðý yönde)
			if(sdx==1)
			{
				//ýþýklara kadar devam et
				if(hdx>=25 && yol10[idx-1*serit_sayi]==0 )
				 {
							sy[id]-=(hiz[id]*7.5);
							yol10[idx-hiz[id]*serit_sayi]=1;
							yol10[idx]=0;
							hcID[id]-=hiz[id];
				 }
		
				//kýrmýzý yandý,dur
				if(TrafficLight[3]==0 && hdx==24)
				 {
					 sy[id]-=(hiz[id]*0.f);
				 }
				 //yeþil yandý, geç
				if(TrafficLight[3]==2 && hdx==24 && yol10[idx-1*serit_sayi]==0)
				 {
					 sy[id]-=(hiz[id]*7.5);
					 yol10[idx-hiz[id]*serit_sayi]=1;
					 yol10[idx]=0;
					 hcID[id]-=(hiz[id]);
				 }
				 //ýþýklarý geçtikten sonra durma ,devam et
				if(hdx<=23 && yol10[idx-1*serit_sayi]==0)
				 {					
						    sy[id]-=(hiz[id]*7.5);
							yol10[idx-hiz[id]*serit_sayi]=1;
							yol10[idx]=0;
							hcID[id]-=(hiz[id]);
				 }

				//yol10-serit0 sonu----->yol2'ye dön
				if(hdx==1)
				{
					
					sx[id]=136*3.75f;
					sy[id]=6*3.75;
					yolID[id]=2;
					srtID[id]=1;
					hcID[id]=68;
					yol10[0*serit_sayi+1]=0;
					yol2[68*serit_sayi+1]=1;
					yon[id]=2;
				}
			}

		}//end of YOL10

		////////////////////////////////////////////////////////
		////----------------ÇAPRAZ YOLLAR----------------------
		if(yolID[id]==104)
		{
			int idx=hcID[id];
			
			
			//kavþak noktasýna kadar ilerle
			
				if(kol4[idx+1]==0)
				{   
					sx[id]+=7.5;
					sy[id]+=7.5;
					kol4[idx+1]=1;
					kol4[idx]=0;
					hcID[id]++;
				}
				if(idx==5)
				{
					sx[id]=108*3.75;
					sy[id]=46*3.75;
					srtID[id]=0;
					yolID[id]=8;
					hcID[id]=54;
					yon[id]=1;
					kol4[5]=0;
					yol8[54*3+0]=1;
				}
			
		}
		//////////////////////////////
		if(yolID[id]==103)
		{
			int idx=hcID[id];
			//kavþak noktasýna kadar ilerle
			
				if(kol3[idx+1]==0)
				{   
					sx[id]+=7.5;
					sy[id]-=7.5;
					kol3[idx+1]=1;
					kol3[idx]=0;
					hcID[id]++;
				}
				if(hcID[id]==5)
				{
					sx[id]=94*3.75;
					sy[id]=36*3.75;
					srtID[id]=1;
					yolID[id]=52;
					hcID[id]=15;
					yon[id]=4;
					kol3[5]=0;
					yol52[15*2+0]=1;
				}
			
		}


		//çapraz yol 204
		if(yolID[id]==204)
		{
			int idx=hcID[id];
			//kavþak noktasýna kadar ilerle
			
				if(kol2_4[idx+1]==0)
				{   
					sx[id]+=7.5;
					sy[id]+=7.5;
					kol2_4[idx+1]=1;
					kol2_4[idx]=0;
					hcID[id]++;
				}

				if(hcID[id]==3)
				{
					sx[id]=148*3.75;
					sy[id]=46*3.75;
					srtID[id]=0;
					yolID[id]=8;
					hcID[id]=74;
					yon[id]=1;
					kol2_4[3]=0;
					yol8[74*3+0]=1;
				}
			
		}
		//çapraz yol 203
		if(yolID[id]==203)
		{
			int idx=hcID[id];
			//kavþak noktasýna kadar ilerle
			
				if(kol2_3[idx+1]==0)
				{   
					sx[id]+=7.5;
					sy[id]-=7.5;
					kol2_3[idx+1]=1;
					kol2_3[idx]=0;
					hcID[id]++;
				}

				if(hcID[id]==2)
				{
					sx[id]=137*3.75;
					sy[id]=42*3.75;
					srtID[id]=1;
					yolID[id]=9;
					hcID[id]=18;
					yon[id]=4;
					kol2_3[2]=0;
					yol9[18*9+1]=1;
				}
			
		}
		//çapraz yol 202
		if(yolID[id]==202)
		{
			int idx=hcID[id];
			//kavþak noktasýna kadar ilerle
			
				if(kol2_2[idx+1]==0)
				{   
					sx[id]-=7.5;
					sy[id]+=7.5;
					kol2_2[idx+1]=1;
					kol2_2[idx]=0;
					hcID[id]++;
				}

				if(hcID[id]==3 )
				{
					sx[id]=141*3.75;
					sy[id]=58*3.75;
					srtID[id]=0;
					yolID[id]=10;
					hcID[id]=27;
					yon[id]=3;
					kol2_2[3]=0;
					yol10[27*2+0]=1;
				}
			
		}
		////////////////
		if(yolID[id]==201)
		{
			int idx=hcID[id];
			//kavþak noktasýna kadar ilerle
			
				if(kol2_1[idx+1]==0)
				{   
					sx[id]-=7.5;
					sy[id]-=7.5;
					kol2_1[idx+1]=1;
					kol2_1[idx]=0;
					hcID[id]++;
				}
				if(hcID[id]==3)
				{
					sx[id]=133*3.75;
					sy[id]=52*3.75;
					srtID[id]=0;
					yolID[id]=7;
					hcID[id]=67;
					yon[id]=2;
					kol2_1[5]=0;
					yol8[54*3+0]=1;
				}
			
		}

}

///////////////////////////////////////////////////
void Initial()
{   
	for(int id=0;id<arac_sayisi;id++)
	{
		SOLX[id]=arac_dizisi[id][0];
		SOLY[id]=arac_dizisi[id][1];
		TIP[id]=arac_dizisi[id][2];
		HIZ[id]=arac_dizisi[id][3];
		YON[id]=arac_dizisi[id][4];
		YOLID[id]=arac_dizisi[id][5];
		SERITID[id]=arac_dizisi[id][6];
		HCID[id]=arac_dizisi[id][7];
		
		
		HucreDoldur(YOLID[id],SERITID[id],HCID[id],TIP[id]);
		
	}//end of for

}//end of initial()
///////////////////////////////////////////////////
void waitAndRedraw(int deger)
{  
		
	    if(animate)
		{
			hipEvent_t start, stop;//****// 
			float zaman;//****// 
			hipEventCreate(&start);//****// 
			hipEventCreate(&stop);//****//

			TL_Time_Depend();
			dim3 threads,blocks;
			threads.x=arac_sayisi;
			blocks.x=1;
			float *dev_SOLX,*dev_SOLY;
			int *dev_TIP,*dev_HIZ,*dev_YON,*dev_YOLID,*dev_SERITID,*dev_HCID;
			int *dev_yol1,*dev_yol2,*dev_yol3,*dev_yol5,*dev_yol52,*dev_yol7;
			int *dev_yol8,*dev_yol9,*dev_yol10;
			int *dev_TL;
			int *dev_kol1,*dev_kol2,*dev_kol3,*dev_kol4;
			int *dev_kol2_1,*dev_kol2_2,*dev_kol2_3,*dev_kol2_4;

			hipEventRecord(start, 0);//****//
			//yer ayýr
			hipMalloc((void**)&dev_SOLX,arac_sayisi*sizeof(float));
			hipMalloc((void**)&dev_SOLY,arac_sayisi*sizeof(float));
			hipMalloc((void**)&dev_TIP,arac_sayisi*sizeof(int));
			hipMalloc((void**)&dev_HIZ,arac_sayisi*sizeof(int));
			hipMalloc((void**)&dev_YON,arac_sayisi*sizeof(int));
			hipMalloc((void**)&dev_YOLID,arac_sayisi*sizeof(int));
			hipMalloc((void**)&dev_SERITID,arac_sayisi*sizeof(int));
			hipMalloc((void**)&dev_HCID,arac_sayisi*sizeof(int));

			hipMalloc((void**)&dev_yol1,35*2*sizeof(int));
			hipMalloc((void**)&dev_yol2,71*2*sizeof(int));
			hipMalloc((void**)&dev_yol3,20*2*sizeof(int));
			hipMalloc((void**)&dev_yol5,45*2*sizeof(int));
			hipMalloc((void**)&dev_yol52,91*3*sizeof(int));
			hipMalloc((void**)&dev_yol7,91*3*sizeof(int));
			hipMalloc((void**)&dev_yol8,91*3*sizeof(int));
			hipMalloc((void**)&dev_yol9,42*2*sizeof(int));
			hipMalloc((void**)&dev_yol10,43*2*sizeof(int));
			hipMalloc((void**)&dev_kol1,6*sizeof(int));
			hipMalloc((void**)&dev_kol2,6*sizeof(int));
			hipMalloc((void**)&dev_kol3,6*sizeof(int));
			hipMalloc((void**)&dev_kol4,6*sizeof(int));
			hipMalloc((void**)&dev_kol2_1,5*sizeof(int));
			hipMalloc((void**)&dev_kol2_2,5*sizeof(int));
			hipMalloc((void**)&dev_kol2_3,5*sizeof(int));
			hipMalloc((void**)&dev_kol2_4,5*sizeof(int));
			

			hipMalloc((void**)&dev_TL,4*sizeof(int));
			/////////////////////////////////////////////////////////////////
			hipMemcpy(dev_SOLX,SOLX,arac_sayisi*sizeof(float),hipMemcpyHostToDevice);
			hipMemcpy(dev_SOLY,SOLY,arac_sayisi*sizeof(float),hipMemcpyHostToDevice);
			hipMemcpy(dev_TIP,TIP,arac_sayisi*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_HIZ,HIZ,arac_sayisi*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_YON,YON,arac_sayisi*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_YOLID,YOLID,arac_sayisi*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_SERITID,SERITID,arac_sayisi*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_HCID,HCID,arac_sayisi*sizeof(int),hipMemcpyHostToDevice);
			
			hipMemcpy(dev_yol1,yol1,35*2*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_yol2,yol2,71*2*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_yol3,yol3,20*2*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_yol5,yol5,45*2*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_yol52,yol52,15*2*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_yol7,yol7,91*3*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_yol8,yol8,91*3*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_yol9,yol9,42*2*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_yol10,yol10,43*2*sizeof(int),hipMemcpyHostToDevice);

			hipMemcpy(dev_kol1,kol1,6*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_kol2,kol2,6*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_kol3,kol3,6*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_kol4,kol4,6*sizeof(int),hipMemcpyHostToDevice);

			hipMemcpy(dev_kol2_1,kol2_1,5*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_kol2_2,kol2_2,5*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_kol2_3,kol2_3,5*sizeof(int),hipMemcpyHostToDevice);
			hipMemcpy(dev_kol2_4,kol2_4,5*sizeof(int),hipMemcpyHostToDevice);

			hipMemcpy(dev_TL,TrafficLight,4*sizeof(int),hipMemcpyHostToDevice);

			
			kernel<<<blocks,threads>>>(dev_SOLX,dev_SOLY,dev_TIP,dev_HIZ,dev_YON,dev_YOLID,dev_SERITID,dev_HCID,dev_yol1,dev_yol2,dev_yol3,dev_yol5,dev_yol52,dev_yol7,dev_yol8,dev_yol9,dev_yol10,dev_TL,dev_kol1,dev_kol2,dev_kol3,dev_kol4,dev_kol2_1,dev_kol2_2,dev_kol2_3,dev_kol2_4);
			
			

			hipMemcpy(SOLX,dev_SOLX,arac_sayisi*sizeof(float),hipMemcpyDeviceToHost);
			hipMemcpy(SOLY,dev_SOLY,arac_sayisi*sizeof(float),hipMemcpyDeviceToHost);
			hipMemcpy(TIP,dev_TIP,arac_sayisi*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(HIZ,dev_HIZ,arac_sayisi*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(YON,dev_YON,arac_sayisi*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(YOLID,dev_YOLID,arac_sayisi*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(SERITID,dev_SERITID,arac_sayisi*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(HCID,dev_HCID,arac_sayisi*sizeof(int),hipMemcpyDeviceToHost);

			
			hipMemcpy(yol1,dev_yol1,35*2*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(yol2,dev_yol2,71*2*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(yol3,dev_yol3,20*2*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(yol5,dev_yol5,45*2*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(yol52,dev_yol52,15*2*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(yol7,dev_yol7,91*3*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(yol8,dev_yol8,91*3*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(yol9,dev_yol9,42*2*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(yol10,dev_yol10,43*2*sizeof(int),hipMemcpyDeviceToHost);
			
			hipMemcpy(kol1,dev_kol1,6*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(kol2,dev_kol2,6*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(kol3,dev_kol3,6*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(kol4,dev_kol4,6*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(kol2_1,dev_kol2_1,5*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(kol2_2,dev_kol2_2,5*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(kol2_3,dev_kol2_3,5*sizeof(int),hipMemcpyDeviceToHost);
			hipMemcpy(kol2_4,dev_kol2_4,5*sizeof(int),hipMemcpyDeviceToHost);

			hipMemcpy(TrafficLight,dev_TL,4*sizeof(int),hipMemcpyDeviceToHost);

		

			hipFree(dev_SOLX);
			hipFree(dev_SOLY);
			hipFree(dev_TIP);
			hipFree(dev_HIZ);
			hipFree(dev_YON);
			hipFree(dev_YOLID);
			hipFree(dev_SERITID);
			hipFree(dev_HCID);

			hipFree(dev_yol1);
			hipFree(dev_yol2);
			hipFree(dev_yol3);
			hipFree(dev_yol5);
			hipFree(dev_yol52);
			hipFree(dev_yol7);
			hipFree(dev_yol8);
			hipFree(dev_yol9);
			hipFree(dev_yol10);
			hipFree(dev_kol1);
			hipFree(dev_kol2);
			hipFree(dev_kol3);
			hipFree(dev_kol4);
			hipFree(dev_TL);
			hipEventRecord(stop, 0); 
			hipEventSynchronize(stop);//****// 
			hipEventElapsedTime(&zaman, start, stop);//****// 

			toplam+=zaman;
			printf ("Time for the Kernel: %f ms-----toplam: %f ms\n", zaman,toplam);//****//
		}
		glutPostRedisplay();
		glutTimerFunc(WAIT,waitAndRedraw,1); 
}
////////////////////////////////////////////////////
void RenderScene(void)
{   
	glClear(GL_COLOR_BUFFER_BIT);
	glPushMatrix();
	
	Harita();
	if(hucreToggle)
	Grid();
	glPopMatrix();
	for(int id=0;id<arac_sayisi;id++)
	{   
		
		araclar[id]=new ARAC(SOLX[id],SOLY[id],TIP[id],HIZ[id],YON[id],YOLID[id],SERITID[id],HCID[id]);
		araclar[id]->AracCiz();
	}
	glutSwapBuffers();
}
///////////////////////////////////////////////////
int main(int argc, char* argv[])
    {    

		  
		  Initial();
          glutInit(&argc, argv);
		  glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB|GLUT_DEPTH );
		  glutInitWindowPosition(0,0);
		  glutInitWindowSize(WINDOW_WIDTH,WINDOW_HEIGHT);
          glutCreateWindow("Hücresel Otomatlar Trafik Simülasyonu(GPU)");
          glutReshapeFunc(ChangeSize);
		  glutTimerFunc(WAIT,waitAndRedraw,1);   
          glutDisplayFunc(RenderScene);
		  glutKeyboardFunc(keyboard);
		  glutMouseFunc(mouse);
          initMenus();
          SetupRC();
          glutMainLoop();
          return 0;
    }

